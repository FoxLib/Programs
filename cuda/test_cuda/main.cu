#include "hip/hip_runtime.h"
#include <stdio.h>
#include <SDL.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = cos(sin(a*x[i]) + y[i]);
}

int main(void)
{
    // 256 млн
    int N = 1 << 27;

    SDL_Init(SDL_INIT_VIDEO | SDL_INIT_TIMER);
    SDL_EnableUNICODE(1);
    SDL_Surface* screen = SDL_SetVideoMode(320, 200, 32, SDL_HWSURFACE | SDL_DOUBLEBUF);
    SDL_WM_SetCaption("CUDA NVCC", 0);

    float *x, *y, *d_x, *d_y;

    // Выделить память на CPU
    x = (float*) malloc(N*sizeof(float));
    y = (float*) malloc(N*sizeof(float));

    // Выделить память на GPU
    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    printf("step1\n");

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    printf("step2\n");

    // Скопировать данные
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    printf("start\n");

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255) / 256, 256>>>(N, 2.0f, d_x, d_y);

    // Подождать пока завершится
    hipDeviceSynchronize();

    printf("stop\n");

    // Скопировать обратно
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    printf(">>\n");

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}

